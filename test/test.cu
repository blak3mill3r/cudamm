#include "hip/hip_runtime.h"
extern "C"
{

texture<float, 2, hipReadModeElementType> inputTexture;
texture<float, 2, hipReadModeElementType> inputTexture2;

__global__ void box_filter(float *out, int width, int height, int pitch)
{
	if(threadIdx.x >= width || threadIdx.y >= height)
	{
		return;
	}

	float val = 0.0;
	for(int i = -1; i <= 1; ++i)
	{
		for(int j = -1; j <= 1; ++j) val += tex2D(inputTexture, threadIdx.x + i, threadIdx.y + j);
	}

	out[threadIdx.y * pitch + threadIdx.x] = val / 9.0;
}

__global__ void difference(float *out, int width, int height, int pitch)
{
	if(threadIdx.x >= width || threadIdx.y >= height)
	{
		return;
	}

	out[threadIdx.y * pitch + threadIdx.x] = 
		max(tex2D(inputTexture2, threadIdx.x, threadIdx.y) - tex2D(inputTexture, threadIdx.x, threadIdx.y)
			, 0.0);
}

}


